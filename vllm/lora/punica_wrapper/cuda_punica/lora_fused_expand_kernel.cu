#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include <cstdio>
#include <cmath>

/**
 * @brief 简化版QKV+LoRA融合expand CUDA kernel
 * 
 * 模仿原始lora_expand_kernel的设计，外部提取好LoRA A数据再传入
 * 执行标准的 output += lora_a_output @ lora_b_weights
 */
template <typename InputT, typename OutputT>
__global__ void lora_fused_qkv_expand_kernel(
    const InputT* __restrict__ fused_matmul_output,  // 融合矩阵乘法的LoRA A输出部分
    const void* __restrict__ lora_b_ptr_array,       
    OutputT* __restrict__ output,                    
    const int* __restrict__ token_indices_sorted,
    const int* __restrict__ lora_ids,
    const int* __restrict__ num_tokens_per_lora,
    const int* __restrict__ lora_token_start_loc,
    const int* __restrict__ slice_starts,            
    const int* __restrict__ lora_a_slice_starts,     
    const int* __restrict__ lora_slice_ranks,        
    const int* __restrict__ lora_strides_d0,         
    const int* __restrict__ lora_strides_d1,
    const int* __restrict__ lora_strides_d2,
    const int* __restrict__ hidden_sizes,            
    int M,                                           
    int MAX_N,                                       
    int qkv_output_size,                             
    int num_slices,
    int max_active_loras,
    bool add_inputs,
    int fused_input_stride0,                         // 融合输入的stride
    int fused_input_stride1,
    int output_stride0,                              
    int output_stride1) {

    // Grid/Block indexing (same as original Punica kernel)
    int cta_m_num = (M + blockDim.y - 1) / blockDim.y;
    int cta_n_num = (MAX_N + blockDim.x - 1) / blockDim.x;
    int pid_mn = blockIdx.x;
    int pid_m = pid_mn % cta_m_num;
    int pid_n = (pid_mn / cta_m_num) % cta_n_num;
    
    int slice_id = blockIdx.y;
    int lora_idx = blockIdx.z; // Index over active LoRAs
    
    // --- Start of Boundary Checks ---
    if (lora_idx >= max_active_loras || slice_id >= num_slices) {
        return;
    }
    
    int lora_id = lora_ids[lora_idx];
    if (lora_id < 0) { // Inactive LoRA slot
        return;
    }
    
    int num_tokens = num_tokens_per_lora[lora_idx];
    int cta_m_offset = pid_m * blockDim.y;
    if (cta_m_offset >= num_tokens) {
        return;
    }
    
    int token_offset = cta_m_offset + threadIdx.y;
    if (token_offset >= num_tokens) {
        return;
    }
    
    int token_start = lora_token_start_loc[lora_idx];
    int actual_token_idx = token_indices_sorted[token_start + token_offset];
    int hidden_idx = pid_n * blockDim.x + threadIdx.x;
    
    int current_slice_hidden_size = hidden_sizes[slice_id];
    
    if (actual_token_idx < 0) {
        return;
    }
    
    if (hidden_idx >= current_slice_hidden_size || actual_token_idx >= M) {
        return;
    }
    // --- End of Boundary Checks ---

    // --- Main Logic ---
    const int64_t* ptr_values = reinterpret_cast<const int64_t*>(lora_b_ptr_array);
    const InputT* cur_lora_b_ptr = reinterpret_cast<const InputT*>(static_cast<uintptr_t>(ptr_values[slice_id]));
    
    // Get strides for the current slice's LoRA B matrix
    int cur_lora_d0_stride = lora_strides_d0[slice_id];
    int cur_lora_d1_stride = lora_strides_d1[slice_id];
    int cur_lora_d2_stride = lora_strides_d2[slice_id];
    
    // Get rank and LoRA A start position for the current (lora_id, slice_id) pair
    int metadata_idx = lora_id * num_slices + slice_id;
    int slice_rank = lora_slice_ranks[metadata_idx];
    if (slice_rank <= 0) {
        return;
    }
    int lora_a_slice_start = lora_a_slice_starts[metadata_idx];

    // Main computation: accumulator = lora_a_output @ lora_b_weights
    float accumulator = 0.0f;
    for (int k = 0; k < slice_rank; k++) {
        // Correctly calculate the index for lora_a_output
        int col = lora_a_slice_start + k;
        int lora_a_offset = actual_token_idx * fused_input_stride0 + col * fused_input_stride1;
        
        // Correctly calculate the index for lora_b_weights
        int lora_b_offset = lora_id * cur_lora_d0_stride + hidden_idx * cur_lora_d1_stride + k * cur_lora_d2_stride;

        // Perform computation
        float lora_a_val = static_cast<float>(fused_matmul_output[lora_a_offset]);
        float lora_b_val = static_cast<float>(cur_lora_b_ptr[lora_b_offset]);
        
        accumulator += lora_a_val * lora_b_val;
    }
    
    // Final check for accumulator to prevent propagating non-finite values.
    if (!isfinite(accumulator)) {
        return;
    }

    // Write result to the output tensor
    int slice_start = slice_starts[slice_id];
    int output_hidden_idx = slice_start + hidden_idx;
    int output_offset = actual_token_idx * output_stride0 + output_hidden_idx * output_stride1;

    // The total size of the output tensor is M * qkv_output_size
    if (output_offset < 0 || output_offset >= M * qkv_output_size) {
        // This should not happen with correct logic, but as a safeguard.
        return;
    }

    if (add_inputs) {
        output[output_offset] += static_cast<OutputT>(accumulator);
    } else {
        output[output_offset] = static_cast<OutputT>(accumulator);
    }
}

/**
 * @brief 简化版LoRA融合expand kernel实现函数
 */
template <typename InputT, typename OutputT>
void lora_fused_qkv_expand_kernel_impl(
    const InputT* fused_matmul_output, const void* lora_b_ptr_array,
    OutputT* output, const int* token_indices_sorted, const int* lora_ids,
    const int* num_tokens_per_lora, const int* lora_token_start_loc,
    const int* slice_starts, const int* lora_a_slice_starts,
    const int* lora_slice_ranks, const int* lora_strides_d0,
    const int* lora_strides_d1, const int* lora_strides_d2,
    const int* hidden_sizes, int max_active_loras, int M, int MAX_N,
    int qkv_output_size, int num_slices, bool add_inputs, int fused_input_stride0,
    int fused_input_stride1, int output_stride0, int output_stride1,
    hipStream_t stream) {

    // --- 安全性检查 ---
    if (max_active_loras <= 0 || num_slices <= 0 || M <= 0 || MAX_N <= 0) {
        // printf("Error: Invalid parameters in fused_qkv_expand_kernel\n");
        return;
    }

    // --- 与原始kernel相同的Grid和Block配置 ---
    const int BLOCK_M = 16;
    const int BLOCK_N = 32;

    int cta_m_num = (M + BLOCK_M - 1) / BLOCK_M;
    int cta_n_num = (MAX_N + BLOCK_N - 1) / BLOCK_N;

    // 定义Grid维度，与原始lora_expand_kernel完全一致
    dim3 grid(cta_m_num * cta_n_num, num_slices, max_active_loras);
    dim3 block(BLOCK_N, BLOCK_M);

    // 启动kernel
    lora_fused_qkv_expand_kernel<InputT, OutputT>
        <<<grid, block, 0, stream>>>(
        fused_matmul_output, lora_b_ptr_array, output,
        token_indices_sorted, lora_ids, num_tokens_per_lora, lora_token_start_loc,
        slice_starts, lora_a_slice_starts, lora_slice_ranks,
        lora_strides_d0, lora_strides_d1, lora_strides_d2, hidden_sizes,
        M, MAX_N, qkv_output_size, num_slices, max_active_loras, add_inputs,
        fused_input_stride0, fused_input_stride1, output_stride0, output_stride1);

    // 检查kernel启动错误
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA fused expand kernel launch error: %s\n", hipGetErrorString(err));
    }
    
    // 等待kernel完成并检查运行时错误
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA fused expand kernel runtime error: %s\n", hipGetErrorString(err));
    }
}

/**
 * @brief 简化版LoRA融合expand kernel启动函数
 */
void launch_lora_fused_expand_kernel(
    const void* fused_matmul_output_ptr,
    const void* lora_b_ptr_array,
    void* output_ptr,
    const int* token_indices_sorted_ptr,
    const int* lora_ids_ptr,
    const int* num_tokens_per_lora_ptr,
    const int* lora_token_start_loc_ptr,
    const int* slice_starts_ptr,
    const int* lora_a_slice_starts_ptr,
    const int* lora_slice_ranks_ptr,
    const int* lora_strides_d0_ptr,
    const int* lora_strides_d1_ptr,
    const int* lora_strides_d2_ptr,
    const int* __restrict__ hidden_sizes_ptr,
    int max_active_loras,
    int num_total_tokens,
    int max_hidden_size,
    int qkv_output_size,
    int num_slices,
    bool add_inputs,
    int fused_input_stride0,
    int fused_input_stride1,
    int output_stride0,
    int output_stride1,
    hipStream_t stream,
    int input_dtype,
    int output_dtype
) {
    // 将输入参数映射到kernel实现函数所需的 M, MAX_N
    int M = num_total_tokens;
    int MAX_N = max_hidden_size;

    // Type dispatch
    if (input_dtype == 1 && output_dtype == 1) { // bf16
        lora_fused_qkv_expand_kernel_impl<__hip_bfloat16, __hip_bfloat16>(
            static_cast<const __hip_bfloat16*>(fused_matmul_output_ptr), lora_b_ptr_array,
            static_cast<__hip_bfloat16*>(output_ptr), token_indices_sorted_ptr, lora_ids_ptr,
            num_tokens_per_lora_ptr, lora_token_start_loc_ptr, slice_starts_ptr,
            lora_a_slice_starts_ptr, lora_slice_ranks_ptr,
            lora_strides_d0_ptr, lora_strides_d1_ptr, lora_strides_d2_ptr,
            hidden_sizes_ptr, max_active_loras, M, MAX_N, qkv_output_size,
            num_slices, add_inputs, fused_input_stride0, fused_input_stride1,
            output_stride0, output_stride1, stream
        );
    } else if (input_dtype == 0 && output_dtype == 0) { // fp16
        lora_fused_qkv_expand_kernel_impl<__half, __half>(
            static_cast<const __half*>(fused_matmul_output_ptr), lora_b_ptr_array,
            static_cast<__half*>(output_ptr), token_indices_sorted_ptr, lora_ids_ptr,
            num_tokens_per_lora_ptr, lora_token_start_loc_ptr, slice_starts_ptr,
            lora_a_slice_starts_ptr, lora_slice_ranks_ptr,
            lora_strides_d0_ptr, lora_strides_d1_ptr, lora_strides_d2_ptr,
            hidden_sizes_ptr, max_active_loras, M, MAX_N, qkv_output_size,
            num_slices, add_inputs, fused_input_stride0, fused_input_stride1,
            output_stride0, output_stride1, stream
        );
    } else if (input_dtype == 2 && output_dtype == 2) { // fp32
        lora_fused_qkv_expand_kernel_impl<float, float>(
            static_cast<const float*>(fused_matmul_output_ptr), lora_b_ptr_array,
            static_cast<float*>(output_ptr), token_indices_sorted_ptr, lora_ids_ptr,
            num_tokens_per_lora_ptr, lora_token_start_loc_ptr, slice_starts_ptr,
            lora_a_slice_starts_ptr, lora_slice_ranks_ptr,
            lora_strides_d0_ptr, lora_strides_d1_ptr, lora_strides_d2_ptr,
            hidden_sizes_ptr, max_active_loras, M, MAX_N, qkv_output_size,
            num_slices, add_inputs, fused_input_stride0, fused_input_stride1,
            output_stride0, output_stride1, stream
        );
    } else {
        printf("Error: Unsupported dtype combination in fused_qkv_expand_kernel. Input: %d, Output: %d\n", 
               input_dtype, output_dtype);
    }
} 

